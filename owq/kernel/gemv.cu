#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "owq_cuda.h"

__global__ void VecQuant3MatMulKernel(
    const    float* __restrict__ vec,
    const      int* __restrict__ mat,
             float* __restrict__ mul,
    const    float* __restrict__ scales,
    const  uint8_t* __restrict__ zeros,
    int height,
    int width
) {
  int row = BLOCKHEIGHT * blockIdx.x;
  int col =  BLOCKWIDTH * blockIdx.y + threadIdx.x;
  int bwidth = ((height - row) < BLOCKHEIGHT) ? ((height - row) * 32 / 3) : BLOCKWIDTH;

  __shared__ float blockvec[BLOCKWIDTH];
  if (threadIdx.x < bwidth)
    blockvec[threadIdx.x] = vec[(row / BLOCKHEIGHT) * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  if (col < width){
    float scale = scales[col];
    float zero = threadIdx.x % 2 ? \
                 float(zeros[col / 2] >> 4) * scale: \
                 float(zeros[col / 2] & 0xf) * scale;

    float res = 0;
    int i = width * row + col;
    int k = 0;

    unsigned int tmp1;
    unsigned int tmp2;
    unsigned int tmp;

    while (k < bwidth) {
      tmp1 = as_unsigned(mat[i]);
      res += (scale * float((tmp1 >>  0) & 0x7) - zero) * blockvec[k + 0];
      res += (scale * float((tmp1 >>  3) & 0x7) - zero) * blockvec[k + 1];
      res += (scale * float((tmp1 >>  6) & 0x7) - zero) * blockvec[k + 2];
      res += (scale * float((tmp1 >>  9) & 0x7) - zero) * blockvec[k + 3];
      res += (scale * float((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
      res += (scale * float((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
      res += (scale * float((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
      res += (scale * float((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
      res += (scale * float((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
      res += (scale * float((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];
      i += width;
      tmp2 = as_unsigned(mat[i]);
      tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x4);
      tmp2 >>= 1;
      res += (scale * float(tmp) - zero) * blockvec[k + 10];
      k += 11;
      res += (scale * float((tmp2 >>  0) & 0x7) - zero) * blockvec[k + 0];
      res += (scale * float((tmp2 >>  3) & 0x7) - zero) * blockvec[k + 1];
      res += (scale * float((tmp2 >>  6) & 0x7) - zero) * blockvec[k + 2];
      res += (scale * float((tmp2 >>  9) & 0x7) - zero) * blockvec[k + 3];
      res += (scale * float((tmp2 >> 12) & 0x7) - zero) * blockvec[k + 4];
      res += (scale * float((tmp2 >> 15) & 0x7) - zero) * blockvec[k + 5];
      res += (scale * float((tmp2 >> 18) & 0x7) - zero) * blockvec[k + 6];
      res += (scale * float((tmp2 >> 21) & 0x7) - zero) * blockvec[k + 7];
      res += (scale * float((tmp2 >> 24) & 0x7) - zero) * blockvec[k + 8];
      res += (scale * float((tmp2 >> 27) & 0x7) - zero) * blockvec[k + 9];
      i += width;
      tmp1 = as_unsigned(mat[i]);
      tmp = (tmp2 >> 30) | ((tmp1 << 1) & 0x6);
      tmp1 >>= 2;
      res += (scale * float(tmp) - zero) * blockvec[k + 10];
      k += 11;
      res += (scale * float((tmp1 >>  0) & 0x7) - zero) * blockvec[k + 0];
      res += (scale * float((tmp1 >>  3) & 0x7) - zero) * blockvec[k + 1];
      res += (scale * float((tmp1 >>  6) & 0x7) - zero) * blockvec[k + 2];
      res += (scale * float((tmp1 >>  9) & 0x7) - zero) * blockvec[k + 3];
      res += (scale * float((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
      res += (scale * float((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
      res += (scale * float((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
      res += (scale * float((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
      res += (scale * float((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
      res += (scale * float((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];
      i += width;
      k += 10;
    }
    atomicAdd(&mul[col], res);
  }
}

template <typename T1, typename T2>
__global__ void VecQuant3MatMulKernelFaster(
    const      T2* __restrict__ vec,
    const     int* __restrict__ mat,
               T2* __restrict__ mul,
    const      T1* __restrict__ scales,
    const uint8_t* __restrict__ zeros,
    int height,
    int width
) {
  const int blockwidth2 = BLOCKWIDTH / 2;

  int row = BLOCKHEIGHT * blockIdx.x;
  int col = BLOCKWIDTH * blockIdx.y + threadIdx.x;
  int bwidth = ((height - row) < BLOCKHEIGHT) ? ((height - row) * 16 / 3) : blockwidth2;

  __shared__ T2 blockvec[blockwidth2];
  if (threadIdx.x < bwidth)
    blockvec[threadIdx.x] = vec[(row / BLOCKHEIGHT) * blockwidth2 + threadIdx.x];

  __shared__ T2 deq2[64][32];
  int val = threadIdx.x / 32;
  int off = threadIdx.x % 32;
  for (; val < 64; val += BLOCKWIDTH / 32) {
    deq2[val][off] = pair2pack(
       int2T<T1>(val & 0x7), int2T<T1>(val >> 3)
    );
  }

  __syncthreads();

  if (col < width) {
    T2 scale = TtoT2(scales[col]);
    T2 zero = threadIdx.x % 2 ? \
              TtoT2(hmul(int2T<T1>(zeros[col / 2] >> 4), hneg(scale.x))) : \
              TtoT2(hmul(int2T<T1>(zeros[col / 2] & 0xf), hneg(scale.x)));

    int i = width * row + col;
    int k = 0;

    float res = 0;
    T2 res2;

    unsigned int tmp1;
    unsigned int tmp2;
    unsigned int tmp;

    while (k < bwidth) {
      res2 = {};
      tmp1 = as_unsigned(mat[i]);
      res2 = hfma2(hfma2(deq2[(tmp1 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >> 24) & 0x3f][off], scale, zero), blockvec[k + 4], res2);
      i += width;
      tmp2 = as_unsigned(mat[i]);
      tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x3c);
      res2 = hfma2(hfma2(deq2[tmp][off], scale, zero), blockvec[k + 5], res2);
      tmp2 >>= 4;
      k += 6;
      res2 = hfma2(hfma2(deq2[(tmp2 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
      res2 = hfma2(hfma2(deq2[(tmp2 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
      res2 = hfma2(hfma2(deq2[(tmp2 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
      res2 = hfma2(hfma2(deq2[(tmp2 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
      i += width;
      tmp1 = as_unsigned(mat[i]);
      tmp = (tmp2 >> 24) | ((tmp1 << 4) & 0x30);
      res2 = hfma2(hfma2(deq2[tmp][off], scale, zero), blockvec[k + 4], res2);
      tmp1 >>= 2;
      k += 5;
      res2 = hfma2(hfma2(deq2[(tmp1 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >> 24) & 0x3f][off], scale, zero), blockvec[k + 4], res2);
      i += width;
      k += 5;
      res += T2float(res2.x) + T2float(res2.y);
    }

    if (col % 2 == 0){
      res2 = pair2pack(float2T<T1>(res), getzero<T1>());
    }
    else{
      res2 = pair2pack(getzero<T1>(), float2T<T1>(res));
    }
    atomicAdd(&mul[col / 2], res2);
  }
}

__global__ void VecQuant3OutlierMatMulKernel(
    const    float* __restrict__ vec,
    const      int* __restrict__ mat,
             float* __restrict__ mul,
    const    float* __restrict__ scales,
    const  uint8_t* __restrict__ zeros,
    const    float* __restrict__ outlierMat,
    const      int* __restrict__ outlieridx,
    const      int* __restrict__ outrow,
    const      int* __restrict__ cnt,
    int height,
    int width
) {
  int row = BLOCKHEIGHT * blockIdx.x;
  int col =  BLOCKWIDTH * blockIdx.y + threadIdx.x;
  int bwidth = ((height - row) < BLOCKHEIGHT) ? ((height - row) * 32 / 3) : BLOCKWIDTH;

  int oidx = -1;
  int blockoutrow = outrow[blockIdx.x];
  int blockcnt = cnt[blockIdx.x];

  outlierMat += blockoutrow * width;
  outlieridx += blockoutrow;

  for (int i = 0; i < blockcnt; i++){
    if (threadIdx.x == outlieridx[i] % BLOCKWIDTH)
      oidx = i;
  }
  
  __shared__ float blockvec[BLOCKWIDTH];
  __shared__ float blockveco[MAXOUTLIER];

  if (threadIdx.x < bwidth){
    blockvec[threadIdx.x] = vec[(row / BLOCKHEIGHT) * BLOCKWIDTH + threadIdx.x];
    if (oidx > -1)
      blockveco[oidx] = blockvec[threadIdx.x];
  }

  __syncthreads();

  if (col < width){
    float scale = scales[col];
    float zero = threadIdx.x % 2 ? \
                 float(zeros[col / 2] >> 4) * scale: \
                 float(zeros[col / 2] & 0xf) * scale;

    float res = 0;
    int i = width * row + col;
    int k = 0;

    unsigned int tmp1;
    unsigned int tmp2;
    unsigned int tmp;

    while (k < bwidth) {
      tmp1 = as_unsigned(mat[i]);
      res += (scale * float((tmp1 >>  0) & 0x7) - zero) * blockvec[k + 0];
      res += (scale * float((tmp1 >>  3) & 0x7) - zero) * blockvec[k + 1];
      res += (scale * float((tmp1 >>  6) & 0x7) - zero) * blockvec[k + 2];
      res += (scale * float((tmp1 >>  9) & 0x7) - zero) * blockvec[k + 3];
      res += (scale * float((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
      res += (scale * float((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
      res += (scale * float((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
      res += (scale * float((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
      res += (scale * float((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
      res += (scale * float((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];
      i += width;
      tmp2 = as_unsigned(mat[i]);
      tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x4);
      tmp2 >>= 1;
      res += (scale * float(tmp) - zero) * blockvec[k + 10];
      k += 11;
      res += (scale * float((tmp2 >>  0) & 0x7) - zero) * blockvec[k + 0];
      res += (scale * float((tmp2 >>  3) & 0x7) - zero) * blockvec[k + 1];
      res += (scale * float((tmp2 >>  6) & 0x7) - zero) * blockvec[k + 2];
      res += (scale * float((tmp2 >>  9) & 0x7) - zero) * blockvec[k + 3];
      res += (scale * float((tmp2 >> 12) & 0x7) - zero) * blockvec[k + 4];
      res += (scale * float((tmp2 >> 15) & 0x7) - zero) * blockvec[k + 5];
      res += (scale * float((tmp2 >> 18) & 0x7) - zero) * blockvec[k + 6];
      res += (scale * float((tmp2 >> 21) & 0x7) - zero) * blockvec[k + 7];
      res += (scale * float((tmp2 >> 24) & 0x7) - zero) * blockvec[k + 8];
      res += (scale * float((tmp2 >> 27) & 0x7) - zero) * blockvec[k + 9];
      i += width;
      tmp1 = as_unsigned(mat[i]);
      tmp = (tmp2 >> 30) | ((tmp1 << 1) & 0x6);
      tmp1 >>= 2;
      res += (scale * float(tmp) - zero) * blockvec[k + 10];
      k += 11;
      res += (scale * float((tmp1 >>  0) & 0x7) - zero) * blockvec[k + 0];
      res += (scale * float((tmp1 >>  3) & 0x7) - zero) * blockvec[k + 1];
      res += (scale * float((tmp1 >>  6) & 0x7) - zero) * blockvec[k + 2];
      res += (scale * float((tmp1 >>  9) & 0x7) - zero) * blockvec[k + 3];
      res += (scale * float((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
      res += (scale * float((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
      res += (scale * float((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
      res += (scale * float((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
      res += (scale * float((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
      res += (scale * float((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];
      i += width;
      k += 10;
    }

    if (blockcnt > 0){
      for (int k = 0; k < blockcnt; k++) {
        res += outlierMat[col + k * width] * blockveco[k];
      }
    }
    atomicAdd(&mul[col], res);
  }
}

template <typename T1, typename T2>
__global__ void VecQuant3OutlierMatMulKernelFaster(
    const       T2* __restrict__ vec,
    const      int* __restrict__ mat,
                T2* __restrict__ mul,
    const       T1* __restrict__ scales,
    const  uint8_t* __restrict__ zeros,
    const       T1* __restrict__ outlierMat,
    const      int* __restrict__ outlieridx,
    const      int* __restrict__ outrow,
    const      int* __restrict__ cnt,
    int height,
    int width
) {
  const int blockwidth2 = BLOCKWIDTH / 2;

  int row = BLOCKHEIGHT * blockIdx.x;
  int col = BLOCKWIDTH * blockIdx.y + threadIdx.x;
  int bwidth = ((height - row) < BLOCKHEIGHT) ? ((height - row) * 16 / 3) : blockwidth2;

  __shared__ T2 deq2[64][32];
  int val = threadIdx.x / 32;
  int off = threadIdx.x % 32;
  for (; val < 64; val += BLOCKWIDTH / 32) {
    deq2[val][off] = pair2pack(
       int2T<T1>(val & 0x7), int2T<T1>(val >> 3)
    );
  }
  
  int flag = 0, oidx = 0;
  int blockoutrow = outrow[blockIdx.x];
  int blockcnt = cnt[blockIdx.x];

  outlierMat += blockoutrow * width;
  outlieridx += blockoutrow;

  for (int i = 0, outidx = 0; i < blockcnt; i++){
    outidx = outlieridx[i];
    if (threadIdx.x == (outidx / 2) % blockwidth2){
      flag += (outidx % 2) + 1; // 1 (x) 2 (y) 3(x,y)
      oidx = i;
    }
  }
  
  __shared__ T2 blockvec[blockwidth2];
  __shared__ T1 blockveco[MAXOUTLIER];

  if (threadIdx.x < bwidth){
    blockvec[threadIdx.x] = vec[(row / BLOCKHEIGHT) * blockwidth2 + threadIdx.x];
    if (flag == 1)
      blockveco[oidx] = blockvec[threadIdx.x].x;
    else if (flag == 2)
      blockveco[oidx] = blockvec[threadIdx.x].y;
    else if (flag == 3){
      blockveco[oidx - 1] = blockvec[threadIdx.x].x;
      blockveco[oidx] = blockvec[threadIdx.x].y;
    }
  }

  __syncthreads();

  if (col < width) {
    T2 scale = TtoT2(scales[col]);
    T2 zero = threadIdx.x % 2 ? \
              TtoT2(hmul(int2T<T1>(zeros[col / 2] >> 4), hneg(scale.x))) : \
              TtoT2(hmul(int2T<T1>(zeros[col / 2] & 0xf), hneg(scale.x)));

    int i = width * row + col;
    int k = 0;

    float res = 0;
    T2 res2;

    unsigned int tmp1;
    unsigned int tmp2;
    unsigned int tmp;

    while (k < bwidth) {
      res2 = {};
      tmp1 = as_unsigned(mat[i]);
      res2 = hfma2(hfma2(deq2[(tmp1 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >> 24) & 0x3f][off], scale, zero), blockvec[k + 4], res2);
      i += width;
      tmp2 = as_unsigned(mat[i]);
      tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x3c);
      res2 = hfma2(hfma2(deq2[tmp][off], scale, zero), blockvec[k + 5], res2);
      tmp2 >>= 4;
      k += 6;
      res2 = hfma2(hfma2(deq2[(tmp2 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
      res2 = hfma2(hfma2(deq2[(tmp2 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
      res2 = hfma2(hfma2(deq2[(tmp2 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
      res2 = hfma2(hfma2(deq2[(tmp2 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
      i += width;
      tmp1 = as_unsigned(mat[i]);
      tmp = (tmp2 >> 24) | ((tmp1 << 4) & 0x30);
      res2 = hfma2(hfma2(deq2[tmp][off], scale, zero), blockvec[k + 4], res2);
      tmp1 >>= 2;
      k += 5;
      res2 = hfma2(hfma2(deq2[(tmp1 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
      res2 = hfma2(hfma2(deq2[(tmp1 >> 24) & 0x3f][off], scale, zero), blockvec[k + 4], res2);
      i += width;
      k += 5;
      res += T2float(res2.x) + T2float(res2.y);
    }

    if (blockcnt > 0){
      T1 res_o = {};
      for (int k = 0; k < blockcnt; k++) {
        res_o = hfma(outlierMat[col + k * width], blockveco[k], res_o);
      }
      res += T2float(res_o);
    }

    if (col % 2 == 0){
      res2 = pair2pack(float2T<T1>(res), getzero<T1>());
    }
    else{
      res2 = pair2pack(getzero<T1>(), float2T<T1>(res));
    }
    atomicAdd(&mul[col / 2], res2);
  }
}

__global__ void VecQuant4MatMulKernel(
    const    float* __restrict__ vec,
    const      int* __restrict__ mat,
             float* __restrict__ mul,
    const    float* __restrict__ scales,
    const  uint8_t* __restrict__ zeros,
    int height,
    int width
) {
  int row = BLOCKHEIGHT4B * blockIdx.x;
  int col =  BLOCKWIDTH * blockIdx.y + threadIdx.x;
  int bwidth = ((height - row) < BLOCKHEIGHT4B) ? ((height - row) * 8) : BLOCKWIDTH;

  __shared__ float blockvec[BLOCKWIDTH];
  if (threadIdx.x < bwidth)
    blockvec[threadIdx.x] = vec[(row / BLOCKHEIGHT4B) * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  if (col < width){
    float scale = scales[col];
    float zero = threadIdx.x % 2 ? \
                 float(zeros[col / 2] >> 4) * scale: \
                 float(zeros[col / 2] & 0xf) * scale;

    float res = 0;
    int i = width * row + col;
    int k = 0;

    unsigned int tmp;

    while (k < bwidth) {
      tmp = as_unsigned(mat[i]);
      for (int a = 0; a < 8; a++){
        res += (scale * float((tmp >> (a * 4)) & 0xf) - zero) * blockvec[k + a];
      }
      i += width;
      k += 8;
    }
    atomicAdd(&mul[col], res);
  }
}

template <typename T1, typename T2>
__global__ void VecQuant4MatMulKernelFaster(
    const       T2* __restrict__ vec,
    const      int* __restrict__ mat,
                T2* __restrict__ mul,
    const       T1* __restrict__ scales,
    const  uint8_t* __restrict__ zeros,
    int height,
    int width
) {
  const int blockwidth2 = BLOCKWIDTH / 2;

  int row = BLOCKHEIGHT4B * blockIdx.x;
  int col = BLOCKWIDTH * blockIdx.y + threadIdx.x;
  int bwidth = ((height - row) < BLOCKHEIGHT4B) ? ((height - row) * 4) : blockwidth2;

  __shared__ T2 blockvec[blockwidth2];
  if (threadIdx.x < bwidth)
    blockvec[threadIdx.x] = vec[(row / BLOCKHEIGHT4B) * blockwidth2 + threadIdx.x];

  __syncthreads();

  if (col < width) {
    T2 scale = TtoT2(scales[col]);
    T2 zero = threadIdx.x % 2 ? \
              TtoT2(hmul(int2T<T1>(zeros[col / 2] >> 4), hneg(scale.x))) : \
              TtoT2(hmul(int2T<T1>(zeros[col / 2] & 0xf), hneg(scale.x)));

    int i = width * row + col;
    int k = 0;

    float res = 0;
    T2 temp;
    T2 res2;

    unsigned int tmp;

    while (k < bwidth) {
      tmp = as_unsigned(mat[i]);
      res2 = {};
      for (int a = 0; a < 4; a++){
        temp = pair2pack(
          int2T<T1>((tmp >> (a * 8)) & 0x0f),
          int2T<T1>((tmp >> (a * 8 + 4)) & 0x0f)
        );
        res2 = hfma2(hfma2(temp, scale, zero), blockvec[k + a], res2);
      }
      i += width;
      k += 4;
      res += T2float(res2.x) + T2float(res2.y);
    }
    if (col % 2 == 0){
      res2 = pair2pack(float2T<T1>(res), getzero<T1>());
    }
    else{
      res2 = pair2pack(getzero<T1>(), float2T<T1>(res));
    }
    atomicAdd(&mul[col / 2], res2);
  }
}

__global__ void VecQuant4OutlierMatMulKernel(
    const    float* __restrict__ vec,
    const      int* __restrict__ mat,
             float* __restrict__ mul,
    const    float* __restrict__ scales,
    const  uint8_t* __restrict__ zeros,
    const    float* __restrict__ outlierMat,
    const      int* __restrict__ outlieridx,
    const      int* __restrict__ outrow,
    const      int* __restrict__ cnt,
    int height,
    int width
) {
  int row = BLOCKHEIGHT4B * blockIdx.x;
  int col =  BLOCKWIDTH * blockIdx.y + threadIdx.x;
  int bwidth = ((height - row) < BLOCKHEIGHT4B) ? ((height - row) * 8) : BLOCKWIDTH;

  int oidx = -1;
  int blockoutrow = outrow[blockIdx.x];
  int blockcnt = cnt[blockIdx.x];

  outlierMat += blockoutrow * width;
  outlieridx += blockoutrow;

  for (int i = 0; i < blockcnt; i++){
    if (threadIdx.x == outlieridx[i] % BLOCKWIDTH)
      oidx = i;
  }
  
  __shared__ float blockvec[BLOCKWIDTH];
  __shared__ float blockveco[MAXOUTLIER];

  if (threadIdx.x < bwidth){
    blockvec[threadIdx.x] = vec[(row / BLOCKHEIGHT4B) * BLOCKWIDTH + threadIdx.x];
    if (oidx > -1)
      blockveco[oidx] = blockvec[threadIdx.x];
  }

  __syncthreads();

  if (col < width){
    float scale = scales[col];
    float zero = threadIdx.x % 2 ? \
                 float(zeros[col / 2] >> 4) * scale: \
                 float(zeros[col / 2] & 0xf) * scale;

    float res = 0;
    int i = width * row + col;
    int k = 0;

    unsigned int tmp;

    while (k < bwidth) {
      tmp = as_unsigned(mat[i]);
      for (int a = 0; a < 8; a++){
        res += (scale * float((tmp >> (a * 4)) & 0xf) - zero) * blockvec[k + a];
      }
      i += width;
      k += 8;
    }
    
    if (blockcnt > 0){
      for (int k = 0; k < blockcnt; k++) {
        res += outlierMat[col + k * width] * blockveco[k];
      }
    }
    atomicAdd(&mul[col], res);
  }
}

template <typename T1, typename T2>
__global__ void VecQuant4OutlierMatMulKernelFaster(
    const       T2* __restrict__ vec,
    const      int* __restrict__ mat,
                T2* __restrict__ mul,
    const       T1* __restrict__ scales,
    const  uint8_t* __restrict__ zeros,
    const       T1* __restrict__ outlierMat,
    const      int* __restrict__ outlieridx,
    const      int* __restrict__ outrow,
    const      int* __restrict__ cnt,
    int height,
    int width
) {
  const int blockwidth2 = BLOCKWIDTH / 2;

  int row = BLOCKHEIGHT4B * blockIdx.x;
  int col = BLOCKWIDTH * blockIdx.y + threadIdx.x;
  int bwidth = ((height - row) < BLOCKHEIGHT4B) ? ((height - row) * 4) : blockwidth2;

  int flag = 0, oidx = 0;
  int blockoutrow = outrow[blockIdx.x];
  int blockcnt = cnt[blockIdx.x];

  outlierMat += blockoutrow * width;
  outlieridx += blockoutrow;

  for (int i = 0, outidx = 0; i < blockcnt; i++){
    outidx = outlieridx[i];
    if (threadIdx.x == (outidx / 2) % blockwidth2){
      flag += (outidx % 2) + 1; // 1 (x) 2 (y) 3(x,y)
      oidx = i;
    }
  }
  
  __shared__ T2 blockvec[blockwidth2];
  __shared__ T1 blockveco[MAXOUTLIER];

  if (threadIdx.x < bwidth){
    blockvec[threadIdx.x] = vec[(row / BLOCKHEIGHT4B) * blockwidth2 + threadIdx.x];
    if (flag == 1)
      blockveco[oidx] = blockvec[threadIdx.x].x;
    else if (flag == 2)
      blockveco[oidx] = blockvec[threadIdx.x].y;
    else if (flag == 3){
      blockveco[oidx - 1] = blockvec[threadIdx.x].x;
      blockveco[oidx] = blockvec[threadIdx.x].y;
    }
  }
  __syncthreads();

  if (col < width) {
    T2 scale = TtoT2(scales[col]);
    T2 zero = threadIdx.x % 2 ? \
              TtoT2(hmul(int2T<T1>(zeros[col / 2] >> 4), hneg(scale.x))) : \
              TtoT2(hmul(int2T<T1>(zeros[col / 2] & 0xf), hneg(scale.x)));

    int i = width * row + col;
    int k = 0;

    float res = 0;
    T2 temp;
    T2 res2;

    unsigned int tmp;

    while (k < bwidth) {
      tmp = as_unsigned(mat[i]);
      res2 = {};
      for (int a = 0; a < 4; a++){
        temp = pair2pack(
          int2T<T1>((tmp >> (a * 8)) & 0x0f),
          int2T<T1>((tmp >> (a * 8 + 4)) & 0x0f)
        );
        res2 = hfma2(hfma2(temp, scale, zero), blockvec[k + a], res2);
      }
      i += width;
      k += 4;
      res += T2float(res2.x) + T2float(res2.y);
    }

    if (blockcnt > 0){
      T1 res_o = {};
      for (int k = 0; k < blockcnt; k++) {
        res_o = hfma(outlierMat[col + k * width], blockveco[k], res_o);
      }
      res += T2float(res_o);
    }

    if (col % 2 == 0){
      res2 = pair2pack(float2T<T1>(res), getzero<T1>());
    }
    else{
      res2 = pair2pack(getzero<T1>(), float2T<T1>(res));
    }
    atomicAdd(&mul[col / 2], res2);
  }

}

void vecquant3matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT - 1) / BLOCKHEIGHT,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant3matmul_cuda", ([&] {
      VecQuant3MatMulKernel<<<blocks, threads>>>(
        vec.data<float>(), mat.data<int>(), mul.data<float>(),
        scales.data<float>(), zeros.data<uint8_t>(),
        height, width
      );
    })
  );
}

void vecquant3matmul_faster_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT - 1) / BLOCKHEIGHT,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);
  if (scales.dtype() == torch::kBFloat16){
    VecQuant3MatMulKernelFaster<<<blocks, threads>>>(
      (nv_bfloat162*) vec.data_ptr(),
      mat.data_ptr<int>(),
      (nv_bfloat162*) mul.data_ptr(),
      (hip_bfloat16*) scales.data_ptr(),
      zeros.data_ptr<uint8_t>(),
      height, width
    );
  }
  else {
    VecQuant3MatMulKernelFaster<<<blocks, threads>>>(
      (half2*) vec.data_ptr(),
      mat.data_ptr<int>(),
      (half2*) mul.data_ptr(),
      (half*) scales.data_ptr(),
      zeros.data_ptr<uint8_t>(),
      height, width
    );
  }
}

void vecquant3outliermatmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  torch::Tensor outlierMat,
  torch::Tensor outlieridx,
  torch::Tensor outrow,
  torch::Tensor cnt
) {
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT - 1) / BLOCKHEIGHT,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant3OutlierMatMulKernel<<<blocks, threads>>>(
    vec.data_ptr<float>(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    scales.data_ptr<float>(),
    zeros.data_ptr<uint8_t>(),
    outlierMat.data_ptr<float>(),
    outlieridx.data_ptr<int>(), 
    outrow.data_ptr<int>(), 
    cnt.data_ptr<int>(),
    height, width
  );
}

void vecquant3outliermatmul_faster_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  torch::Tensor outlierMat,
  torch::Tensor outlieridx,
  torch::Tensor outrow,
  torch::Tensor cnt
) {
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT - 1) / BLOCKHEIGHT,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  if (scales.dtype() == torch::kBFloat16){
    VecQuant3OutlierMatMulKernelFaster<<<blocks, threads>>>(
      (nv_bfloat162*) vec.data_ptr(),
      mat.data_ptr<int>(),
      (nv_bfloat162*) mul.data_ptr(),
      (hip_bfloat16*) scales.data_ptr(),
      zeros.data_ptr<uint8_t>(),
      (hip_bfloat16*) outlierMat.data_ptr(), 
      outlieridx.data_ptr<int>(), 
      outrow.data_ptr<int>(), 
      cnt.data_ptr<int>(), 
      height, width
    );
  }
  else {
    VecQuant3OutlierMatMulKernelFaster<<<blocks, threads>>>(
      (half2*) vec.data_ptr(),
      mat.data_ptr<int>(),
      (half2*) mul.data_ptr(),
      (half*) scales.data_ptr(),
      zeros.data_ptr<uint8_t>(),
      (half*) outlierMat.data_ptr(), 
      outlieridx.data_ptr<int>(), 
      outrow.data_ptr<int>(), 
      cnt.data_ptr<int>(), 
      height, width
    );
  }
}

void vecquant4matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4B - 1) / BLOCKHEIGHT4B,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant4matmul_cuda", ([&] {
      VecQuant4MatMulKernel<<<blocks, threads>>>(
        vec.data<float>(), mat.data<int>(), mul.data<float>(),
        scales.data<float>(), zeros.data<uint8_t>(),
        height, width
      );
    })
  );
}

void vecquant4matmul_faster_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4B - 1) / BLOCKHEIGHT4B,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  if (scales.dtype() == torch::kBFloat16){
    VecQuant4MatMulKernelFaster<<<blocks, threads>>>(
      (nv_bfloat162*) vec.data_ptr(),
      mat.data_ptr<int>(),
      (nv_bfloat162*) mul.data_ptr(),
      (hip_bfloat16*) scales.data_ptr(),
      zeros.data_ptr<uint8_t>(),
      height, width
    );
  }
  else{
    VecQuant4MatMulKernelFaster<<<blocks, threads>>>(
      (half2*) vec.data_ptr(),
      mat.data_ptr<int>(),
      (half2*) mul.data_ptr(),
      (half*) scales.data_ptr(),
      zeros.data_ptr<uint8_t>(),
      height, width
    );
  }
}

void vecquant4outliermatmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  torch::Tensor outlierMat,
  torch::Tensor outlieridx,
  torch::Tensor outrow,
  torch::Tensor cnt
) {
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4B - 1) / BLOCKHEIGHT4B,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant4OutlierMatMulKernel<<<blocks, threads>>>(
    vec.data_ptr<float>(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    scales.data_ptr<float>(),
    zeros.data_ptr<uint8_t>(),
    outlierMat.data_ptr<float>(),
    outlieridx.data_ptr<int>(), 
    outrow.data_ptr<int>(), 
    cnt.data_ptr<int>(),
    height, width
  );
}

void vecquant4outliermatmul_faster_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  torch::Tensor outlierMat,
  torch::Tensor outlieridx,
  torch::Tensor outrow,
  torch::Tensor cnt
) {
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4B - 1) / BLOCKHEIGHT4B,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);
  
  if (scales.dtype() == torch::kBFloat16){
    VecQuant4OutlierMatMulKernelFaster<<<blocks, threads>>>(
      (nv_bfloat162*) vec.data_ptr(),
      mat.data_ptr<int>(),
      (nv_bfloat162*) mul.data_ptr(),
      (hip_bfloat16*) scales.data_ptr(),
      zeros.data_ptr<uint8_t>(),
      (hip_bfloat16*) outlierMat.data_ptr(), 
      outlieridx.data_ptr<int>(), 
      outrow.data_ptr<int>(), 
      cnt.data_ptr<int>(), 
      height, width
    );
  }
  else { // Half
    VecQuant4OutlierMatMulKernelFaster<<<blocks, threads>>>(
      (half2*) vec.data_ptr(),
      mat.data_ptr<int>(),
      (half2*) mul.data_ptr(),
      (half*) scales.data_ptr(),
      zeros.data_ptr<uint8_t>(),
      (half*) outlierMat.data_ptr(), 
      outlieridx.data_ptr<int>(), 
      outrow.data_ptr<int>(), 
      cnt.data_ptr<int>(), 
      height, width
    );
  }
}
